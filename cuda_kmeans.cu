#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "kmeans.h"

static inline int nextPowerOfTwo(int n) {
    n--;

    n = n >>  1 | n;
    n = n >>  2 | n;
    n = n >>  4 | n;
    n = n >>  8 | n;
    n = n >> 16 | n;
//  n = n >> 32 | n;    //  64-bits

    return ++n;
}

//distancia euclidea
__host__ __device__ inline static
float euclid_dist_2(int    numCoords,
                    int    numObjs,
                    int    numClusters,
                    float *objects,     // [numCoords][numObjs]
                    float *clusters,    // [numCoords][numClusters]
                    int    objectId,
                    int    clusterId)
{
    int i;
    float ans=0.0;

    for (i = 0; i < numCoords; i++) {
        ans += (objects[numObjs * i + objectId] - clusters[numClusters * i + clusterId]) *
               (objects[numObjs * i + objectId] - clusters[numClusters * i + clusterId]);
    }

    return(ans);
}

// encuentra el cluster más cercano
__global__ static
void find_nearest_cluster(int numCoords,
                          int numObjs,
                          int numClusters,
                          float *objects,           //  [numCoords][numObjs]
                          float *deviceClusters,    //  [numCoords][numClusters]
                          int *membership,          //  [numObjs]
                          int *intermediates)
{
    extern __shared__ char sharedMemory[];

    // Hay elementos blockDim.x, uno para cada hilo en el bloque
    unsigned char *membershipChanged = (unsigned char *)sharedMemory;
#if BLOCK_SHARED_MEM_OPTIMIZATION
    float *clusters = (float *)(sharedMemory + blockDim.x);
#else
    float *clusters = deviceClusters;
#endif

    membershipChanged[threadIdx.x] = 0;

#if BLOCK_SHARED_MEM_OPTIMIZATION

    //  Se puede sobrecargar la memoria compartida si hay muchos clusters o muchas coordenadas

    for (int i = threadIdx.x; i < numClusters; i += blockDim.x) {
        for (int j = 0; j < numCoords; j++) {
            clusters[numClusters * j + i] = deviceClusters[numClusters * j + i];
        }
    }
    __syncthreads();
#endif

    int objectId = blockDim.x * blockIdx.x + threadIdx.x;

    if (objectId < numObjs) {
        int   index, i;
        float dist, min_dist;

        /* encuentra el id del cluster más cercano */
        index    = 0;
        min_dist = euclid_dist_2(numCoords, numObjs, numClusters, objects, clusters, objectId, 0);

        for (i=1; i<numClusters; i++)
	{
            dist = euclid_dist_2(numCoords, numObjs, numClusters, objects, clusters, objectId, i);
            if (dist < min_dist)
	    {
                min_dist = dist;
                index    = i;
            }
        }

        if (membership[objectId] != index) {
            membershipChanged[threadIdx.x] = 1;
        }

        /* asigna el id al objeto */
        membership[objectId] = index;

        __syncthreads(); 

        for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
            if (threadIdx.x < s) {
                membershipChanged[threadIdx.x] +=
                    membershipChanged[threadIdx.x + s];
            }
            __syncthreads();
        }

        if (threadIdx.x == 0) {
            intermediates[blockIdx.x] = membershipChanged[0];
        }
    }
}

__global__ static
void compute_delta(int *deviceIntermediates, int numIntermediates, int numIntermediates2)  
{

    extern __shared__ unsigned int intermediates[];

    intermediates[threadIdx.x] =
        (threadIdx.x < numIntermediates) ? deviceIntermediates[threadIdx.x] : 0;

    __syncthreads();

    for (unsigned int s = numIntermediates2 / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            intermediates[threadIdx.x] += intermediates[threadIdx.x + s];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        deviceIntermediates[0] = intermediates[0];
    }
}

//  DATA LAYOUT
//
//  objects         [numObjs][numCoords]
//  clusters        [numClusters][numCoords]
//  dimObjects      [numCoords][numObjs]
//  dimClusters     [numCoords][numClusters]
//  newClusters     [numCoords][numClusters]
//  deviceObjects   [numCoords][numObjs]
//  deviceClusters  [numCoords][numClusters]
//  ----------------------------------------
//
/* retorna un array de centro de clusters de tamaño [numClusters][numCoords]       */
float** cuda_kmeans(float **objects,      /* entrada: [numObjs][numCoords] */
                   int     numCoords,    /* num caracteristicas */
                   int     numObjs,      /* num objetos */
                   int     numClusters,  /* num clusters */
                   float   threshold, 	
                   int    *membership,   /* salida: [numObjs] */
                   int    *loop_iterations)
{
    int      i, j, index, loop=0;
    int     *newClusterSize; /* [numClusters]: num de objetos nuevos en el cluster */
    float    delta;          
    float  **dimObjects;
    float  **clusters;       /* salida: [numClusters][numCoords] */
    float  **dimClusters;
    float  **newClusters;    /* [numCoords][numClusters] */

    float *deviceObjects;
    float *deviceClusters;
    int *deviceMembership;
    int *deviceIntermediates;

    malloc2D(dimObjects, numCoords, numObjs, float);
    for (i = 0; i < numCoords; i++) {
        for (j = 0; j < numObjs; j++) {
            dimObjects[i][j] = objects[j][i];
        }
    }

    malloc2D(dimClusters, numCoords, numClusters, float);
    for (i = 0; i < numCoords; i++) {
        for (j = 0; j < numClusters; j++) {
            dimClusters[i][j] = dimObjects[i][j];
        }
    }

    for (i=0; i<numObjs; i++) membership[i] = -1;

    newClusterSize = (int*) calloc(numClusters, sizeof(int));
    assert(newClusterSize != NULL);

    malloc2D(newClusters, numCoords, numClusters, float);
    memset(newClusters[0], 0, numCoords * numClusters * sizeof(float));

    const unsigned int numThreadsPerClusterBlock = 128;
    const unsigned int numClusterBlocks =
        (numObjs + numThreadsPerClusterBlock - 1) / numThreadsPerClusterBlock;
#if BLOCK_SHARED_MEM_OPTIMIZATION
    const unsigned int clusterBlockSharedDataSize =
        numThreadsPerClusterBlock * sizeof(unsigned char) +
        numClusters * numCoords * sizeof(float);

    hipDeviceProp_t deviceProp;
    int deviceNum;
    hipGetDevice(&deviceNum);
    hipGetDeviceProperties(&deviceProp, deviceNum);

    if (clusterBlockSharedDataSize > deviceProp.sharedMemPerBlock) {
        err("Insuficiente memoria compartida para los bloques\n");
    }
#else
    const unsigned int clusterBlockSharedDataSize =
        numThreadsPerClusterBlock * sizeof(unsigned char);
#endif

    const unsigned int numReductionThreads =
        nextPowerOfTwo(numClusterBlocks);
    const unsigned int reductionBlockSharedDataSize =
        numReductionThreads * sizeof(unsigned int);

    checkCuda(hipMalloc(&deviceObjects, numObjs*numCoords*sizeof(float)));
    checkCuda(hipMalloc(&deviceClusters, numClusters*numCoords*sizeof(float)));
    checkCuda(hipMalloc(&deviceMembership, numObjs*sizeof(int)));
    checkCuda(hipMalloc(&deviceIntermediates, numReductionThreads*sizeof(unsigned int)));

  // copiamos los datos de los puntos y los centros aleatorios en el dispositivo

    checkCuda(hipMemcpy(deviceObjects, dimObjects[0], numObjs*numCoords*sizeof(float), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(deviceMembership, membership, numObjs*sizeof(int), hipMemcpyHostToDevice));

    do {
        checkCuda(hipMemcpy(deviceClusters, dimClusters[0],
                  numClusters*numCoords*sizeof(float), hipMemcpyHostToDevice));

  // se inicializa el Kernel con: Num de bloques por cluster, num de hilos por bloque de cluster,
  // tamaño de datos de los bloqes de cluster compartidos

        find_nearest_cluster
            <<< numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize >>>
            (numCoords, numObjs, numClusters,
             deviceObjects, deviceClusters, deviceMembership, deviceIntermediates);

        hipDeviceSynchronize(); checkLastCudaError();

        compute_delta <<< 1, numReductionThreads, reductionBlockSharedDataSize >>>
            (deviceIntermediates, numClusterBlocks, numReductionThreads);
// se espera a que el dispositivo termine de procesar los datos
        hipDeviceSynchronize(); checkLastCudaError();

// se copian los resultados en el host

        int d;
        checkCuda(hipMemcpy(&d, deviceIntermediates, sizeof(int), hipMemcpyDeviceToHost));
        delta = (float)d;

        checkCuda(hipMemcpy(membership, deviceMembership, numObjs*sizeof(int), hipMemcpyDeviceToHost));

        for (i=0; i<numObjs; i++) {
            /* encontrar el indice del cetro del cluster */
            index = membership[i];

            /* actualizar los nuevos centros */
            newClusterSize[index]++;
            for (j=0; j<numCoords; j++)
                newClusters[j][index] += objects[i][j];
        }

        //  TODO: Cambiar el orden de los ppuntos
        //  TODO: [numClusters][numCoords]
        /* suma promedio y reemplazar los antiguos clusters*/
        for (i=0; i<numClusters; i++) {
            for (j=0; j<numCoords; j++) {
                if (newClusterSize[i] > 0)
                    dimClusters[j][i] = newClusters[j][i] / newClusterSize[i];
                newClusters[j][i] = 0.0;
            }
            newClusterSize[i] = 0;
        }

        delta /= numObjs;
    } while (delta > threshold && loop++ < 500);

    *loop_iterations = loop + 1;

    // asignar un espacio 2d para las coordenadas de los clusters

    malloc2D(clusters, numClusters, numCoords, float);
    for (i = 0; i < numClusters; i++) {
        for (j = 0; j < numCoords; j++) {
            clusters[i][j] = dimClusters[j][i];
        }
    }

    checkCuda(hipFree(deviceObjects));
    checkCuda(hipFree(deviceClusters));
    checkCuda(hipFree(deviceMembership));
    checkCuda(hipFree(deviceIntermediates));

    free(dimObjects[0]);
    free(dimObjects);
    free(dimClusters[0]);
    free(dimClusters);
    free(newClusters[0]);
    free(newClusters);
    free(newClusterSize);

    return clusters;
}

